#include <iostream>
#include <cmath>
#include <chrono>
#include <hip/hip_runtime.h>
#include <hipDNN.h>

#define C 3
#define H 1024
#define W 1024
#define K 64
#define FW 3
#define FH 3
#define P 1

void generate_input(double *I, double *F) {
    for (int c = 0; c < C; ++c) {
        for (int x = 0; x < H; ++x) {
            for (int y = 0; y < W; ++y) {
                I[c * H * W + x * W + y] = c * (x + y);
            }
        }
    }

    for (int k = 0; k < K; ++k) {
        for (int c = 0; c < C; ++c) {
            for (int i = 0; i < FH; ++i) {
                for (int j = 0; j < FW; ++j) {
                    F[k * C * FH * FW + c * FH * FW + i * FW + j] = (c + k) * (i + j);
                }
            }
        }
    }
}

int main() {
    double *I = new double[C * H * W];
    double *F = new double[K * C * FH * FW];
    double *O = new double[K * H * W];

    generate_input(I, F);

    hipdnnHandle_t cudnn;
    hipdnnCreate(&cudnn);

    hipdnnTensorDescriptor_t input_descriptor;
    hipdnnCreateTensorDescriptor(&input_descriptor);
    hipdnnSetTensor4dDescriptor(input_descriptor, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_DOUBLE, 1, C, H, W);

    hipdnnFilterDescriptor_t kernel_descriptor;
    hipdnnCreateFilterDescriptor(&kernel_descriptor);
    hipdnnSetFilter4dDescriptor(kernel_descriptor, HIPDNN_DATA_DOUBLE, HIPDNN_TENSOR_NCHW, K, C, FH, FW);

    hipdnnConvolutionDescriptor_t convolution_descriptor;
    hipdnnCreateConvolutionDescriptor(&convolution_descriptor);
    hipdnnSetConvolution2dDescriptor(convolution_descriptor, P, P, 1, 1, 1, 1, HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_DOUBLE);

    int out_n, out_c, out_h, out_w;
    hipdnnGetConvolution2dForwardOutputDim(convolution_descriptor, input_descriptor, kernel_descriptor, &out_n, &out_c, &out_h, &out_w);

    hipdnnTensorDescriptor_t output_descriptor;
    hipdnnCreateTensorDescriptor(&output_descriptor);
    hipdnnSetTensor4dDescriptor(output_descriptor, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_DOUBLE, out_n, out_c, out_h, out_w);

    hipdnnConvolutionFwdAlgo_t convolution_algorithm;
    hipdnnGetConvolutionForwardAlgorithm(cudnn, input_descriptor, kernel_descriptor, convolution_descriptor, output_descriptor, HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, &convolution_algorithm);

    size_t workspace_size;
    hipdnnGetConvolutionForwardWorkspaceSize(cudnn, input_descriptor, kernel_descriptor, convolution_descriptor, output_descriptor, convolution_algorithm, &workspace_size);

    void *workspace;
    hipMalloc(&workspace, workspace_size);

    double *d_input, *d_kernel, *d_output;
    hipMalloc((void **)&d_input, C * H * W * sizeof(double));
    hipMalloc((void **)&d_kernel, K * C * FH * FW * sizeof(double));
    hipMalloc((void **)&d_output, K * H * W * sizeof(double));

    hipMemcpy(d_input, I, C * H * W * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_kernel, F, K * C * FH * FW * sizeof(double), hipMemcpyHostToDevice);

    double alpha = 1.0;
    double beta = 0.0;

    auto start = std::chrono::high_resolution_clock::now();

    hipdnnConvolutionForward(cudnn, &alpha, input_descriptor, d_input, kernel_descriptor, d_kernel, convolution_descriptor, convolution_algorithm, workspace, workspace_size, &beta, output_descriptor, d_output);

    hipDeviceSynchronize();

    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed = end - start;
    std::cout << "Convolution execution time: " << elapsed.count() << " seconds" << std::endl;

    hipMemcpy(O, d_output, K * H * W * sizeof(double), hipMemcpyDeviceToHost);

    double checksum = 0;
    for (int i = 0; i < K * H * W; ++i) {
        checksum += O[i];
    }
    std::cout << "Checksum: " << checksum << std::endl;

    hipFree(workspace);
    hipFree(d_input);
    hipFree(d_kernel);
    hipFree(d_output);

    hipdnnDestroyTensorDescriptor(input_descriptor);
    hipdnnDestroyTensorDescriptor(output_descriptor);
    hipdnnDestroyFilterDescriptor(kernel_descriptor);
    hipdnnDestroyConvolutionDescriptor(convolution_descriptor);
    hipdnnDestroy(cudnn);

    delete[] I;
    delete[] F;
    delete[] O;

    return 0;
}