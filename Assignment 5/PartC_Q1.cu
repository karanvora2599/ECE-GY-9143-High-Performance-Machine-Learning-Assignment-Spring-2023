#include <iostream>
#include <chrono>
#include <hip/hip_runtime.h>

#define H 1024
#define W 1024
#define C 3
#define FW 3
#define FH 3
#define K 64

__global__ void simple_convolution(const double *I0, const double *F, double *O) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z;

    if (x < W && y < H) {
        double sum = 0;
        for (int c = 0; c < C; c++) {
            for (int i = 0; i < FW; i++) {
                for (int j = 0; j < FH; j++) {
                    sum += F[k * C * FW * FH + c * FW * FH + (FW - 1 - i) * FH + (FH - 1 - j)] * I0[c * (W + 2) * (H + 2) + (x + i) * (H + 2) + (y + j)];
                }
            }
        }
        O[k * W * H + x * H + y] = sum;
    }
}

int main() {
    double *I, *F, *I0, *O;
    double *d_I0, *d_F, *d_O;
    size_t input_size = C * H * W * sizeof(double);
    size_t filter_size = K * C * FH * FW * sizeof(double);
    size_t padded_input_size = C * (W + 2) * (H + 2) * sizeof(double);
    size_t output_size = K * H * W * sizeof(double);

    // Allocate memory for the input, filter, and output tensors on the CPU
    hipHostMalloc((void **)&I, input_size, hipHostMallocDefault);
    hipHostMalloc((void **)&F, filter_size, hipHostMallocDefault);
    hipHostMalloc((void **)&I0, padded_input_size, hipHostMallocDefault);
    hipHostMalloc((void **)&O, output_size, hipHostMallocDefault);

    // Generate the input tensor I and filter F
    for (int c = 0; c < C; c++) {
        for (int x = 0; x < H; x++) {
            for (int y = 0; y < W; y++) {
                I[c * H * W + x * W + y] = c * (x + y);
            }
        }
    }

    for (int k = 0; k < K; k++) {
        for (int c = 0; c < C; c++) {
            for (int i = 0; i < FH; i++) {
                for (int j = 0; j < FW; j++) {
                    F[k * C * FH * FW + c * FH * FW + i * FW + j] = (c + k) * (i + j);
                }
            }
                    }
    }

    // Generate the padded input tensor I0
    for (int c = 0; c < C; c++) {
        for (int x = 0; x < W + 2; x++) {
            for (int y = 0; y < H + 2; y++) {
                if (x == 0 || x == W + 1 || y == 0 || y == H + 1) {
                    I0[c * (W + 2) * (H + 2) + x * (H + 2) + y] = 0;
                } else {
                    I0[c * (W + 2) * (H + 2) + x * (H + 2) + y] = I[c * H * W + (x - 1) * W + (y - 1)];
                }
            }
        }
    }

    // Allocate memory for the input, filter, and output tensors on the GPU
    hipMalloc((void **)&d_I0, padded_input_size);
    hipMalloc((void **)&d_F, filter_size);
    hipMalloc((void **)&d_O, output_size);

    // Copy the input tensor and filter to the GPU
    hipMemcpy(d_I0, I0, padded_input_size, hipMemcpyHostToDevice);
    hipMemcpy(d_F, F, filter_size, hipMemcpyHostToDevice);

    // Set the kernel's execution configuration
    dim3 blockDim(16, 16);
    dim3 gridDim((W + blockDim.x - 1) / blockDim.x, (H + blockDim.y - 1) / blockDim.y, K);

    // Launch the kernel and measure the execution time
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    simple_convolution<<<gridDim, blockDim>>>(d_I0, d_F, d_O);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "Kernel execution time: " << milliseconds << " ms" << std::endl;

    // Copy the output tensor back to the CPU
    hipMemcpy(O, d_O, output_size, hipMemcpyDeviceToHost);

    // Calculate the checksum
    double checksum = 0;
    for (int k = 0; k < K; k++) {
        for (int x = 0; x < W; x++) {
            for (int y = 0; y < H; y++) {
                checksum += O[k * W * H + x * H + y];
            }
        }
    }

    std::cout << "Checksum: " << checksum << std::endl;

    // Free the allocated memory on the GPU and CPU
    hipFree(d_I0);
    hipFree(d_F);
    hipFree(d_O);
    hipHostFree(I);
    hipHostFree(F);
    hipHostFree(I0);
    hipHostFree(O);

    return 0;
}