#include "hip/hip_runtime.h"
// vecadd.cu
// For ECE-GY 9143 - High Performance Computing for Machine Learning
// Instructor: Parijat Dubey
// Based on code from the CUDA Programming Guide

// Add two Vectors A and B in C on GPU using
// a kernel defined according to vecAddKernel.h

// DO NOT MODIFY FOR THE ASSIGNMENT

// Includes
#include <stdio.h>
#include "timer.h"
#include "vecaddKernel.h"

// Defines
#define GridWidth 60
#define BlockWidth 128

// Variables for host and device vectors.
float* h_A; 
float* h_B; 
float* h_C; 
float* d_A; 
float* d_B; 
float* d_C; 

// Utility Functions
void Cleanup(bool);
void checkCUDAError(const char *msg);

// Host code performs setup and calls the kernel.
int main(int argc, char** argv)
{
    int ValuesPerThread; // number of values per thread
    int N; //Vector size

	// Parse arguments.
    if(argc != 2){
     printf("Usage: %s ValuesPerThread\n", argv[0]);
     printf("ValuesPerThread is the number of values added by each thread.\n");
     printf("Total vector size is 128 * 60 * this value.\n");
     exit(0);
    } else {
      sscanf(argv[1], "%d", &ValuesPerThread);
    }      

    // Determine the number of threads .
    // N is the total number of values to be in a vector
    N = ValuesPerThread * GridWidth * BlockWidth;
    printf("Total vector size: %d\n", N); 
    // size_t is the total number of bytes for a vector.
    size_t size = N * sizeof(float);

    // Tell CUDA how big to make the grid and thread blocks.
    // Since this is a vector addition problem,
    // grid and thread block are both one-dimensional.
    dim3 dimGrid(GridWidth);                    
    dim3 dimBlock(BlockWidth);                 

    // Allocate input vectors h_A and h_B in host memory
    h_A = (float*)malloc(size);
    if (h_A == 0) Cleanup(false);
    h_B = (float*)malloc(size);
    if (h_B == 0) Cleanup(false);
    h_C = (float*)malloc(size);
    if (h_C == 0) Cleanup(false);

    // Allocate vectors in device memory.
    hipError_t error;
    error = hipMalloc((void**)&d_A, size);
    if (error != hipSuccess) Cleanup(false);
    error = hipMalloc((void**)&d_B, size);
    if (error != hipSuccess) Cleanup(false);
    error = hipMalloc((void**)&d_C, size);
    if (error != hipSuccess) Cleanup(false);

    // Initialize host vectors h_A and h_B
    int i;
    for(i=0; i<N; ++i){
     h_A[i] = (float)i;
     h_B[i] = (float)(N-i);   
    }

    // Copy host vectors h_A and h_B to device vectores d_A and d_B
    error = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    if (error != hipSuccess) Cleanup(false);
    error = hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
    if (error != hipSuccess) Cleanup(false);

    // Warm up
    AddVectors<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, ValuesPerThread);
    error = hipGetLastError();
    if (error != hipSuccess) Cleanup(false);
    hipDeviceSynchronize();

    // Initialize timer  
    initialize_timer();
    start_timer();

    // Invoke kernel
    AddVectors<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, ValuesPerThread);
    error = hipGetLastError();
    if (error != hipSuccess) Cleanup(false);
    hipDeviceSynchronize();

    // Compute elapsed time 
    stop_timer();
    double time = elapsed_time();

    // Compute floating point operations per second.
    int nFlops = N;
    double nFlopsPerSec = nFlops/time;
    double nGFlopsPerSec = nFlopsPerSec*1e-9;

	// Compute transfer rates.
    int nBytes = 3*4*N; // 2N words in, 1N word out
    double nBytesPerSec = nBytes/time;
    double nGBytesPerSec = nBytesPerSec*1e-9;

	// Report timing data.
    printf( "Time: %lf (sec), GFlopsS: %lf, GBytesS: %lf\n", 
             time, nGFlopsPerSec, nGBytesPerSec);
     
    // Copy result from device memory to host memory
    error = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
    if (error != hipSuccess) Cleanup(false);

    // Verify & report result
    for (i = 0; i < N; ++i) {
        float val = h_C[i];
        if (fabs(val - N) > 1e-5)
            break;
    }
    printf("Test %s \n", (i == N) ? "PASSED" : "FAILED");

    // Clean up and exit.
    Cleanup(true);
}

void Cleanup(bool noError) {  // simplified version from CUDA SDK
    hipError_t error;
        
    // Free device vectors
    if (d_A)
        hipFree(d_A);
    if (d_B)
        hipFree(d_B);
    if (d_C)
        hipFree(d_C);

    // Free host memory
    if (h_A)
        free(h_A);
    if (h_B)
        free(h_B);
    if (h_C)
        free(h_C);
        
    error = hipDeviceReset();
    
    if (!noError || error != hipSuccess)
        printf("cuda malloc or cuda thread exit failed \n");
    
    fflush( stdout);
    fflush( stderr);

    exit(0);
}

void checkCUDAError(const char *msg)
{
  hipError_t err = hipGetLastError();
  if( hipSuccess != err) 
    {
      fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err) );
      exit(-1);
    }                         
}


