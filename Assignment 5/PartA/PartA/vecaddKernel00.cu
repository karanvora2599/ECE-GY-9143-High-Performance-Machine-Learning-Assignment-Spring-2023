
#include <hip/hip_runtime.h>
// vecAddKernel00.cu
// For ECE-GY 9143 - High Performance Computing for Machine Learning
// Instructor: Parijat Dubey
// Based on code from the CUDA Programming Guide

// This Kernel adds two Vectors A and B in C on GPU
// without using coalesced memory access.

__global__ void AddVectors(const float* A, const float* B, float* C, int N)
{
    int blockStartIndex  = blockIdx.x * blockDim.x * N;
    int threadStartIndex = blockStartIndex + (threadIdx.x * N);
    int threadEndIndex   = threadStartIndex + N;
    int i;

    for( i=threadStartIndex; i<threadEndIndex; ++i ){
        C[i] = A[i] + B[i];
    }
}
