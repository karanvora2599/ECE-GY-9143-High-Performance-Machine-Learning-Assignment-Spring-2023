#include "hip/hip_runtime.h"
// matmultKernel00.cu
// For ECE-GY 9143 - High Performance Computing for Machine Learning
// Instructor: Parijat Dubey
// Based on code from the CUDA Programming Guide

// Multiplies two matrices using CUDA: A x B = C

// Copy this file and modify the MatMultKernel device function for
// any of your experiments. 

#include "matmultKernel.h"

#define FOOTPRINT_SIZE BLOCK_SIZE

// Define a gpu kernel to perform matrix multiplication
// of A x B = C.
__global__ void MatMulKernel(Matrix A, Matrix B, Matrix C){

  // matrix blocks
  float *Asub, *Bsub, *Csub;
  // Putting these into registers speeds access.
  int thread_row = threadIdx.y;
  int thread_col = threadIdx.x;
  int block_row = blockIdx.y;
  int block_col = blockIdx.x;

  // Each THREAD BLOCK computes one sub matrix Csub of C
  // EACH THREAD creates its own matrix descriptor Csub
  Csub = &C.elements[C.stride * BLOCK_SIZE * block_row + BLOCK_SIZE * block_col];

  // Each thread computes one element of Csub in its copy of CValue
  float Cvalue = 0;

  // Loop over all sub matrices in block_row of A and block_col of B
  // required to compute Csub. Block multiply each pair of sub matrices
  // and accumulate results
  for (int m = 0;  m < (A.width / BLOCK_SIZE); ++m){
    // Get Asub and Bsub descriptors
    Asub = &A.elements[A.stride * BLOCK_SIZE * block_row + BLOCK_SIZE * m];
    Bsub = &B.elements[B.stride * BLOCK_SIZE * m + BLOCK_SIZE * block_col];

    // Copy ELEMENTS OF  ASub and Bsub into shared memory
    // EACH THREAD loads ONE ELEMENT of ASub and ONE of Bsub
    // Notice: it does not need to be the element it requires to
    //         compute its Cvalue, as long as all elements are 
    //         collaboratively read. 

    // Notice: every thread declares shared_A and shared_B in shared memory
    //         even though a thread block has only one shared_A and one shared_B
    __shared__ float shared_A[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float shared_B[BLOCK_SIZE][BLOCK_SIZE];

    // Each thread copies just one element of shared_A and one element of shared_B
    shared_A[thread_row][thread_col] = Asub[thread_row * A.stride + thread_col];
    shared_B[thread_row][thread_col] = Bsub[thread_row * B.stride + thread_col];

    // Synchronize to ensure all elements are read
    __syncthreads();

    // Do an inproduct of one row of shared_A and one col of shared_B
    // computing one Cvalue by accumulation
#pragma unroll
    for(int e=0; e<BLOCK_SIZE; ++e)
       Cvalue += shared_A[thread_row][e] * shared_B[e][thread_col];

    // Synchronize to ensure all Cvalues have been incremented
    // before reading in the next shared_A AND shared_B BLOCKS
    __syncthreads();
  }

  // Write Csub to GLOBAL memory.
  // Each thread writes its own cell value.
  Csub[thread_row * C.stride + thread_col] = Cvalue;
}

