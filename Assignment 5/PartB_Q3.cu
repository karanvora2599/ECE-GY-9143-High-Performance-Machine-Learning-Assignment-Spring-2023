#include <iostream>
#include <chrono>
#include <hip/hip_runtime.h>

using namespace std;
using namespace chrono;

__global__ void addArrays(int* A, int* B, int* C, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        C[i] = A[i] + B[i];
    }
}

int main() {
    const int K[] = {1, 5, 10, 50, 100};
    const int BLOCK_SIZE = 256;

    for (int k = 0; k < 5; k++) {
        const int n = K[k] * 1000000;

        int* A, *B, *C;
        hipMallocManaged(&A, n * sizeof(int));
        hipMallocManaged(&B, n * sizeof(int));
        hipMallocManaged(&C, n * sizeof(int));

        for (int i = 0; i < n; i++) {
            A[i] = i;
            B[i] = i;
        }

        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        // Scenario 1: using one block with 1 thread
        hipEventRecord(start);
        addArrays<<<1, 1>>>(A, B, C, n);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        float time1;
        hipEventElapsedTime(&time1, start, stop);

        // Scenario 2: using one block with 256 threads
        hipEventRecord(start);
        addArrays<<<1, BLOCK_SIZE>>>(A, B, C, n);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        float time2;
        hipEventElapsedTime(&time2, start, stop);

        // Scenario 3: using multiple blocks with 256 threads per block
        int numBlocks = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;
        hipEventRecord(start);
        addArrays<<<numBlocks, BLOCK_SIZE>>>(A, B, C, n);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        float time3;
        hipEventElapsedTime(&time3, start, stop);

        cout << "K=" << K[k] << endl;
        cout << "Scenario 1: " << time1 << " ms" << endl;
        cout << "Scenario 2: " << time2 << " ms" << endl;
        cout << "Scenario 3: " << time3 << " ms" << endl;

        hipFree(A);
        hipFree(B);
        hipFree(C);
    }

    return 0;
}
